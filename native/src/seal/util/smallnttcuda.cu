#include "hip/hip_runtime.h"
#include "seal/util/smallnttcuda.h"
#include "seal/util/smallntt.h"
#include <stdint.h>


void ntt_negacyclic_harvey_lazy_(uint64_t *operand,
        const uint64_t *root_powers, const uint64_t *scaled_root_powers,
        uint64_t modulus, size_t n){
    size_t t = n >> 1;
    uint64_t *d_operand;
    hipMallocManaged(&d_operand, t*sizeof(uint64_t));
    d_operand = operand;  // Use some form of zerocopy semantics?

    cuda_ntt_negacyclic_harvey_lazy_<<<1, 1>>>(operand, root_powers,
      scaled_root_powers, modulus, n);

    hipDeviceSynchronize();
    hipFree(d_operand);
    // *operand = *d_operand;
}


__global__ void cuda_ntt_negacyclic_harvey_lazy_(
  uint64_t *operand,
  const uint64_t *root_powers, const uint64_t *scaled_root_powers,
  uint64_t modulus, size_t n
){
    uint64_t two_times_modulus = modulus * 2;

    // Return the NTT in scrambled order
    size_t t = n >> 1;
    for (size_t m = 1; m < n; m <<= 1)
    {
        if (t >= 4)
        {
            for (size_t i = 0; i < m; i++)
            {
                size_t j1 = 2 * i * t;
                size_t j2 = j1 + t;
                const uint64_t W = root_powers[m + i];
                const uint64_t Wprime = scaled_root_powers[m + i];

                uint64_t *X = operand + j1;
                uint64_t *Y = X + t;
                uint64_t currX;
                unsigned long long Q;
                for (size_t j = j1; j < j2; j += 4)
                {
                    currX = *X - (two_times_modulus & static_cast<uint64_t>(-static_cast<int64_t>(*X >= two_times_modulus)));
                    multiply_uint64_hw64(Wprime, *Y, &Q);
                    Q = *Y * W - Q * modulus;
                    *X++ = currX + Q;
                    *Y++ = currX + (two_times_modulus - Q);

                    currX = *X - (two_times_modulus & static_cast<uint64_t>(-static_cast<int64_t>(*X >= two_times_modulus)));
                    multiply_uint64_hw64(Wprime, *Y, &Q);
                    Q = *Y * W - Q * modulus;
                    *X++ = currX + Q;
                    *Y++ = currX + (two_times_modulus - Q);

                    currX = *X - (two_times_modulus & static_cast<uint64_t>(-static_cast<int64_t>(*X >= two_times_modulus)));
                    multiply_uint64_hw64(Wprime, *Y, &Q);
                    Q = *Y * W - Q * modulus;
                    *X++ = currX + Q;
                    *Y++ = currX + (two_times_modulus - Q);

                    currX = *X - (two_times_modulus & static_cast<uint64_t>(-static_cast<int64_t>(*X >= two_times_modulus)));
                    multiply_uint64_hw64(Wprime, *Y, &Q);
                    Q = *Y * W - Q * modulus;
                    *X++ = currX + Q;
                    *Y++ = currX + (two_times_modulus - Q);
                }
            }
        }
        else
        {
            for (size_t i = 0; i < m; i++)
            {
                size_t j1 = 2 * i * t;
                size_t j2 = j1 + t;
                const uint64_t W = root_powers[m + i];
                const uint64_t Wprime = scaled_root_powers[m + i];

                uint64_t *X = operand + j1;
                uint64_t *Y = X + t;
                uint64_t currX;
                unsigned long long Q;
                for (size_t j = j1; j < j2; j++)
                {
                    // The Harvey butterfly: assume X, Y in [0, 2p), and return X', Y' in [0, 4p).
                    // X', Y' = X + WY, X - WY (mod p).
                    currX = *X - (two_times_modulus & static_cast<uint64_t>(-static_cast<int64_t>(*X >= two_times_modulus)));
                    multiply_uint64_hw64(Wprime, *Y, &Q);
                    Q = W * *Y - Q * modulus;
                    *X++ = currX + Q;
                    *Y++ = currX + (two_times_modulus - Q);
                }
            }
        }
        t >>= 1;
    }
}

template<typename T, typename S>
__device__ inline void multiply_uint64_hw64(
      T operand1, S operand2, unsigned long long *hw64)
        {
            auto operand1_coeff_right = operand1 & 0x00000000FFFFFFFF;
            auto operand2_coeff_right = operand2 & 0x00000000FFFFFFFF;
            operand1 >>= 32;
            operand2 >>= 32;

            auto middle1 = operand1 * operand2_coeff_right;
            T middle;
            auto left = operand1 * operand2 + (static_cast<T>(add_uint64(
                middle1, operand2 * operand1_coeff_right, &middle)) << 32);
            auto right = operand1_coeff_right * operand2_coeff_right;
            auto temp_sum = (right >> 32) + (middle & 0x00000000FFFFFFFF);

            *hw64 = static_cast<unsigned long long>(
                left + (middle >> 32) + (temp_sum >> 32));
        }

template<typename T, typename S>
__device__ inline unsigned char add_uint64_generic(
        T operand1, S operand2, unsigned char carry,
        unsigned long long *result)
        {
            operand1 += operand2;
            *result = operand1 + carry;
            return (operand1 < operand2) || (~operand1 < carry);
        }

template<typename T, typename S>
__device__ inline unsigned char add_uint64(
    T operand1, S operand2, unsigned char carry,
    unsigned long long *result)
{
    return add_uint64_generic(operand1, operand2, carry, result);
}

template<typename T, typename S, typename R>
__device__ inline unsigned char add_uint64(
    T operand1, S operand2, R *result)
{
    *result = operand1 + operand2;
    return static_cast<unsigned char>(*result < operand1);
}
